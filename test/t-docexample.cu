/*
* This program uses the host CURAND API to generate 100
* pseudorandom floats.
*/

#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand.h>

class RNG {};

class QRNG : public RNG {};

class PRNG : public PRNG {};

#define CUDA_CALL(x) do { if((x) != hipSuccess) {	\
	    printf("Error at %s:%d\n",__FILE__,__LINE__);	\
	    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) {	\
	    printf("Error at %s:%d\n",__FILE__,__LINE__);	\
	    return EXIT_FAILURE;}} while(0)

int main(int argc, char *argv[])
{
    size_t n = 5;
    size_t i;
    hiprandGenerator_t gen;
    float *devData, *hostData;

    /* Allocate n floats on host */
    //hostData = (float *)calloc(n, sizeof(float));

    hostData = new float[n];

    /* Allocate n floats on device */
    CUDA_CALL(hipMalloc((void **)&devData, n * sizeof(float)));

    /* Create pseudo-random number generator */
    // CURAND_CALL(hiprandCreateGenerator(&gen,
    // 				      HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL(hiprandCreateGenerator(&gen,
    				      HIPRAND_RNG_QUASI_SOBOL32));

    /* Set seed */
    // CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

    /* Generate n floats on device */
    CURAND_CALL(hiprandGenerateUniform(gen, devData, n));

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(hostData, devData, n * sizeof(float),
			 hipMemcpyDeviceToHost));

    /* Show result */
    for(i = 0; i < n; i++)
	{
	    std::cout << hostData[i] << " ";
	}
    std::cout << std::endl;

    /* Cleanup */
    CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(devData));
    delete hostData;

    return EXIT_SUCCESS;
}
